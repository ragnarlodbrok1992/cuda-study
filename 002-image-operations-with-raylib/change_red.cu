#include "hip/hip_runtime.h"
#include "cuda_kernels.h"

#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void changeRed() {
  printf("Hello from GPU\n");
}

void changeRed(uint32_t* data, size_t size, int newRed) {
  changeRed<<<1, 1>>>();
  hipDeviceSynchronize();
}
