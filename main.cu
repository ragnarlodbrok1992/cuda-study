#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello() {
  printf("Hello world from the GPU!!!\n");
}

int main(int argc, char* argv[]) {
  cuda_hello<<<1,1>>>();
  hipDeviceSynchronize();
  return 0;
}
